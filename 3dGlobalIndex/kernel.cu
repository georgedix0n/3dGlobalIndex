#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

__global__ void unique_gid_calculation_3d(int *data)
{
	int tid = ((blockDim.x * threadIdx.y) + threadIdx.x) + ((blockDim.x * blockDim.y) * threadIdx.z);


	int blockId = blockIdx.x + (gridDim.x * blockIdx.y) + (gridDim.x * gridDim.y * blockIdx.z);

	int num_threads_in_block = blockId * blockDim.x * blockDim.y;

	int block_offset = num_threads_in_block * blockDim.z;


	int gid = tid + block_offset;
	printf("blockIdx.x : %d, blockIdx.y : %d, blockIdx.z : %d, threadIdx.x : %d, threadIdx.y : %d, threadIdx.z : %d, gid : %d - data :%d \n", blockIdx.x, blockIdx.y, blockIdx.z, threadIdx.x, threadIdx.y, threadIdx.z, gid, data[gid]);

}

int main()
{
	int array_size = 64;
	int array_byte_size = sizeof(int) * array_size;
	int h_data[] = { 780, 484, 58, 722, 359, 255, 173, 696, 209, 562, 161, 57, 37, 262, 398, 266, 845, 864, 337, 197, 510, 961, 124, 990, 753, 84, 673, 183, 204, 966, 708, 939, 772, 28, 98, 211, 53, 471, 803, 498, 697, 416, 763, 588, 950, 776, 404, 819, 452, 14, 487, 203, 390, 205, 387, 550, 219, 794, 974, 490, 538, 913, 13, 251 };

	int* d_data;
	hipMalloc((void**)&d_data, array_byte_size);
	hipMemcpy(d_data, h_data, array_byte_size, hipMemcpyHostToDevice);

	dim3 block(2, 2, 2);
	dim3 grid(2, 2, 2);

	unique_gid_calculation_3d << <grid, block >> > (d_data);
	hipDeviceSynchronize();

	hipDeviceReset();
	return 0;
}